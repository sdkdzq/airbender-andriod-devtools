/*
 * Copyright 2013 NVIDIA Corporation. All rights reserved.
 *
 * Sample CUPTI app to demonstrate the usage of unified memory counter profiling
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cupti.h>

#define CUPTI_CALL(call)                                                    \
do {                                                                        \
    CUptiResult _status = call;                                             \
    if (_status != CUPTI_SUCCESS) {                                         \
      const char *errstr;                                                   \
      cuptiGetResultString(_status, &errstr);                               \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
              __FILE__, __LINE__, #call, errstr);                           \
      exit(-1);                                                             \
    }                                                                       \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define BUF_SIZE (8 * 1024)
#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
    (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))

static const char *
getUvmCounterKindString(CUpti_ActivityUnifiedMemoryCounterKind kind)
{
    switch (kind) 
    {
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_HTOD:
        return "BYTES_TRANSFER_HTOD";
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_DTOH:
        return "BYTES_TRANSFER_DTOH";
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_CPU_PAGE_FAULT_COUNT:
        return "CPU_PAGE_FAULT_COUNT";
    default:
        break;
    }
    return "<unknown>";
}

static const char *
getUvmCounterScopeString(CUpti_ActivityUnifiedMemoryCounterScope scope)
{
    switch (scope) 
    {
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_SCOPE_PROCESS_SINGLE_DEVICE:
        return "PROCESS_SINGLE_DEVICE";
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_SCOPE_PROCESS_ALL_DEVICES:
        return "PROCESS_ALL_DEVICES";
    default:
        break;
    }
    return "<unknown>";
}

static void
printActivity(CUpti_Activity *record)
{
    switch (record->kind) 
    {
    case CUPTI_ACTIVITY_KIND_UNIFIED_MEMORY_COUNTER:
        {
            CUpti_ActivityUnifiedMemoryCounter *uvm = (CUpti_ActivityUnifiedMemoryCounter *)record;
            printf("UNIFIED_MEMORY_COUNTER [ %llu ] scope=%s kind=%s value=%llu device %u\n",
                (unsigned long long)(uvm->timestamp),
                getUvmCounterScopeString(uvm->scope),
                getUvmCounterKindString(uvm->counterKind),
                (unsigned long long)uvm->value,
                uvm->deviceId);
            break;
        }
    default:
        printf("  <unknown>\n");
        break;
    }
}

static void CUPTIAPI
bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)
{
    uint8_t *rawBuffer;

    *size = BUF_SIZE;
    rawBuffer = (uint8_t *)malloc(*size + ALIGN_SIZE);

    *buffer = ALIGN_BUFFER(rawBuffer, ALIGN_SIZE);
    *maxNumRecords = 0;

    if (*buffer == NULL) {
        printf("Error: out of memory\n");
        exit(-1);
    }
}

static void CUPTIAPI
bufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize)
{
    CUptiResult status;

    if (ctx == NULL) {
        printf("==== Starting dump for global ====\n");
    } else if (streamId == 0) {
        printf("==== Starting dump for context %p ====\n", ctx);
    } else {
        printf("==== Starting dump for context %p, stream %u ====\n", ctx, streamId);
    }

    CUpti_Activity *record = NULL;
    do {
        status = cuptiActivityGetNextRecord(buffer, validSize, &record);
        if (status == CUPTI_SUCCESS) {
            printActivity(record);
        }
        else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
            break;
        }
        else {
            CUPTI_CALL(status);
        }
    } while (1);

    // report any records dropped from the queue
    size_t dropped;
    CUPTI_CALL(cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped));
    if (dropped != 0) {
        printf("Dropped %u activity records\n", (unsigned int)dropped);
    }

    if (ctx == NULL) {
        printf("==== Finished dump for global ====\n");
    } else if (streamId == 0) {
        printf("==== Finished dump for context %p ====\n", ctx);
    } else {
        printf("==== Finished dump for context %p, stream %u ====\n", ctx, streamId);
    }

    free(buffer);
}

template<class T>
__host__ __device__ void checkData(const char *loc, T *data, int size, int expectedVal) {
    int i;

    for (i = 0; i < size / (int)sizeof(T); i++) {
        if (data[i] != expectedVal) {
            printf("Mismatch found on %s\n", loc);
            printf("Address 0x%p, Observed = 0x%x Expected = 0x%x\n", data+i, data[i], expectedVal);
            break;
        }
    }
}

template<class T>
__host__ __device__ void writeData(T *data, int size, int writeVal) {
    int i;

    for (i = 0; i < size / (int)sizeof(T); i++) {
        data[i] = writeVal;
    }
}

__global__ void testKernel(int *data, int size, int expectedVal)
{
    checkData("GPU", data, size, expectedVal);
    writeData(data, size, -expectedVal);
}

int main(int argc, char **argv)
{
    CUptiResult res;
    int deviceCount;
    int *data = NULL;
    int size = 64*1024;     // 64 KB
    int i = 123;
    CUpti_ActivityUnifiedMemoryCounterConfig config[3];

    DRIVER_API_CALL(hipInit(0));

    DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(-1);
    }

    // register cupti activity buffer callbacks
    CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));

    // configure unified memory counters
    config[0].scope = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_SCOPE_PROCESS_SINGLE_DEVICE;
    config[0].kind = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_HTOD;
    config[0].deviceId = 0;
    config[0].enable = 1;

    config[1].scope = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_SCOPE_PROCESS_SINGLE_DEVICE;
    config[1].kind = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_DTOH;
    config[1].deviceId = 0;
    config[1].enable = 1;

    config[2].scope = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_SCOPE_PROCESS_SINGLE_DEVICE;
    config[2].kind = CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_CPU_PAGE_FAULT_COUNT;
    config[2].deviceId = 0;
    config[2].enable = 1;

    res = cuptiActivityConfigureUnifiedMemoryCounter(config, 3);
    if (res == CUPTI_ERROR_NOT_SUPPORTED) {
        printf("Test is waived, unified memory is not supported on the underlying platform.\n");
        return 0;
    }
    else {
        CUPTI_CALL(res);
    }

    // enable unified memory counter activity
    CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_UNIFIED_MEMORY_COUNTER));

    // allocate unified memory
    printf("Allocation size in bytes %d\n", size);
    RUNTIME_API_CALL(hipMallocManaged(&data, size));

    // CPU access
    writeData(data, size, i);
    // kernel launch
    testKernel<<<1,1>>>(data, size, i);
    RUNTIME_API_CALL(hipDeviceSynchronize());
    // CPU access
    checkData("CPU", data, size, -i);

    // free unified memory
    RUNTIME_API_CALL(hipFree(data));

    CUPTI_CALL(cuptiActivityFlushAll(0));

    // disable unified memory counter activity
    CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_UNIFIED_MEMORY_COUNTER));

    hipDeviceReset();

    return 0;
}
