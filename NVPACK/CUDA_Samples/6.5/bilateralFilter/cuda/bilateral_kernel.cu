#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "hip/hip_vector_types.h"
#include "helper_functions.h"
#include "bilateral_kernel.h"
#include <android/log.h>

#ifndef _BILATERAL_FILTER_KERNEL_CU_
#define _BILATERAL_FILTER_KERNEL_CU_

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        LOGD("CUDA Runtime API error: %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__constant__ float cGaussian[64];   //gaussian array in device side
texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;

/*
    Perform a simple bilateral filter.

    Bilateral filter is a nonlinear filter that is a mixture of range
    filter and domain filter, the previous one preserves crisp edges and
    the latter one filters noise. The intensity value at each pixel in
    an image is replaced by a weighted average of intensity values from
    nearby pixels.

    The weight factor is calculated by the product of domain filter
    component(using the gaussian distribution as a spatial distance) as
    well as range filter component(Euclidean distance between center pixel
    and the current neighbor pixel). Because this process is nonlinear,
    the sample just uses a simple pixel by pixel step.
    Texture fetches automatically clamp to edge of image. 1D gaussian array
    is mapped to a 1D texture instead of using shared memory, which may
    cause severe bank conflict.

    Threads are y-pass(column-pass), because the output is coalesced.

*/

namespace gpu_bilateral_filter
{

    //Euclidean Distance (x, y, d) = exp((|x - y| / d)^2 / 2)
    __device__ float euclideanLen(float4 a, float4 b, float d)
    {

        float mod = (b.x - a.x) * (b.x - a.x) +
                    (b.y - a.y) * (b.y - a.y) +
                    (b.z - a.z) * (b.z - a.z);

        return __expf(-mod / (2.f * d * d));
    }

    __device__ uint rgbaFloatToInt(float4 rgba)
    {
        rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
        rgba.y = __saturatef(fabs(rgba.y));
        rgba.z = __saturatef(fabs(rgba.z));
        rgba.w = __saturatef(fabs(rgba.w));
        return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
    }

    __device__ float4 rgbaIntToFloat(uint c)
    {
        float4 rgba;
        rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
        rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
        rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
        rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
        return rgba;
    }

    /*
        Bilateral filter kernel
        Parameters
        od - pointer to output data in global memory
        w  - image width
        h  - image height
        euclidean - euclidean delta
        r  - filter radius
    */
    __global__ void d_bilateral_filter(uint *od, int w, int h, float euclidean, int r)
    {
        int x = blockIdx.x*blockDim.x + threadIdx.x;
        int y = blockIdx.y*blockDim.y + threadIdx.y;

        if (x >= w || y >= h)
        {
            return;
        }

        float sum = 0.0f;
        float factor;
        float4 t = {0.f, 0.f, 0.f, 0.f};
        float4 center = tex2D(rgbaTex, x, y);

        for (int i = -r; i <= r; i++)
        {
            for (int j = -r; j <= r; j++)
            {
                float4 curPix = tex2D(rgbaTex, x + j, y + i);
                factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
                         euclideanLen(curPix, center, euclidean);  // range factor
                t +=  factor * curPix;
                sum += factor;
            }
        }

        od[y * w + x] = rgbaFloatToInt(t/sum);
    }

    BilateralFilter::BilateralFilter()
    {
    }

    BilateralFilter::~BilateralFilter()
    {
        // free device memory
        checkCudaErrors(hipFree(d_image));
        checkCudaErrors(hipFree(d_result));
        checkCudaErrors(hipFree(d_temp));
    }

    // Initialize the texture with input image data
    void BilateralFilter::initTexture(int width, int height, uint *hImage)
    {
        // Allocate memory on device
        checkCudaErrors(hipMallocPitch(&d_image, &m_pitch, sizeof(uint)*width, height));
        checkCudaErrors(hipMallocPitch(&d_temp, &m_pitch, sizeof(uint)*width, height));
        checkCudaErrors(hipMallocPitch(&d_result,  &m_pitch, sizeof(uint)*width, height));
        // Copy image data from host memory to device global memory
        checkCudaErrors(hipMemcpy2D(d_image, m_pitch, hImage, sizeof(uint)*width,
                                     sizeof(uint)*width, height, hipMemcpyHostToDevice));
    }

    /*
        Because a 2D gaussian mask is symmetry in row and column,
        here only generate a 1D mask, and use the product by row
        and column index later.

        1D gaussian distribution :
            g(x, d) -- C * exp(-x^2/d^2), C is a constant amplifier

        parameters:
        og - output gaussian array in global memory
        delta - the 2nd parameter 'd' in the above function
        radius - half of the filter size
                 (total filter size = 2 * radius + 1)
    */
    void BilateralFilter::updateGaussian(float delta, int radius)
    {
        float  fGaussian[64] = {0};

        for (int i = 0; i < 2*radius + 1; ++i)
        {
            float x = i-radius;
            fGaussian[i] = expf(-(x*x) / (2*delta*delta));
        }

        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float)*(2*radius+1),
                                           0, hipMemcpyHostToDevice));
    }

    /*
        Perform 2D bilateral filter on image using CUDA

        Parameters:
        hDest - pointer to destination image in host memory
        width  - image width
        height - image height
        euclidean - euclidean delta
        radius - filter radius
        iterations - number of iterations
    */

    // RGBA version
    void BilateralFilter::bilateralFilterRGBA(uint *hDest,
                                              int width,
                                              int height,
                                              float euclidean,
                                              int radius,
                                              int iterations)
    {
        // Bind the array to the texture
        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        checkCudaErrors(hipBindTexture2D(0, rgbaTex, d_image, desc, width, height, m_pitch));

        for (int i=0; i<iterations; i++)
        {
            // Allocate grid size and block size
            dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
            dim3 blockSize(16, 16);

            // Apply simple bilateral filter
            d_bilateral_filter<<< gridSize, blockSize>>> (d_result, width, height, euclidean, radius);

            if (iterations > 1)
            {
                // Copy intermediate result to temp storage in global memory
                checkCudaErrors(hipMemcpy2D(d_temp, m_pitch, d_result, sizeof(int)*width,
                                             sizeof(int)*width, height, hipMemcpyDeviceToDevice));
                // Bind temp result to texture
                checkCudaErrors(hipBindTexture2D(0, rgbaTex, d_temp, desc, width, height, m_pitch));
            }
        }

        // copy back final result to host memory
        checkCudaErrors(hipMemcpy2D(hDest, m_pitch, d_result, sizeof(uint)*width,
                                     sizeof(uint)*width, height, hipMemcpyDeviceToHost));
    }
}
#endif // #ifndef _BILATERAL_FILTER_KERNEL_CU_

