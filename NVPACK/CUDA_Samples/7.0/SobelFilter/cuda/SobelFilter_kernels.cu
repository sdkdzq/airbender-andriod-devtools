#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include "helper_string.h"
#include "SobelFilter_kernels.h"

#include <android/log.h>
#define APP_NAME "CUDA_CORE"

#define LOGD(...) ((void)__android_log_print(ANDROID_LOG_DEBUG,  \
                                             APP_NAME, \
                                             __VA_ARGS__))
// Texture reference for reading image
texture<unsigned char, 2> tex;
extern __shared__ unsigned char LocalBlock[];
static hipArray *array = NULL;

#define RADIUS 1

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

namespace gpu_sobel_filter
{

    // compute sobel on 3x3 element matrix
    __device__ unsigned char
    ComputeSobel(unsigned char ul, // upper left
                 unsigned char um, // upper middle
                 unsigned char ur, // upper right
                 unsigned char ml, // middle left
                 unsigned char mm, // middle (unused)
                 unsigned char mr, // middle right
                 unsigned char ll, // lower left
                 unsigned char lm, // lower middle
                 unsigned char lr, // lower right
                 float fScale)
    {
        short Horz = ur + 2*mr + lr - ul - 2*ml - ll;
        short Vert = ul + 2*um + ur - ll - 2*lm - lr;
        short Sum = (short)(fScale*(abs((int)Horz)+abs((int)Vert)));

        if (Sum < 0)
        {
            return 0;
        }
        else if (Sum > 0xff)
        {
            return 0xff;
        }

        return (unsigned char) Sum;
    }

    // Sobel filter application using shared memory
    __global__ void
    SobelShared(uchar4 *pSobelOriginal, unsigned short SobelPitch,
                short BlockWidth, short SharedPitch,
                short w, short h, float fScale)
    {
        short u = 4*blockIdx.x*BlockWidth;
        short v = blockIdx.y*blockDim.y + threadIdx.y;
        short ib;

        int SharedIdx = threadIdx.y * SharedPitch;

        for (ib = threadIdx.x; ib < BlockWidth+2*RADIUS; ib += blockDim.x)
        {
            LocalBlock[SharedIdx+4*ib+0] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+0), (float)(v-RADIUS));
            LocalBlock[SharedIdx+4*ib+1] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+1), (float)(v-RADIUS));
            LocalBlock[SharedIdx+4*ib+2] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+2), (float)(v-RADIUS));
            LocalBlock[SharedIdx+4*ib+3] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+3), (float)(v-RADIUS));
        }

        if (threadIdx.y < RADIUS*2)
        {
            //
            // copy trailing RADIUS*2 rows of pixels into shared
            //
            SharedIdx = (blockDim.y+threadIdx.y) * SharedPitch;

            for (ib = threadIdx.x; ib < BlockWidth+2*RADIUS; ib += blockDim.x)
            {
                LocalBlock[SharedIdx+4*ib+0] = tex2D(tex,
                                                     (float)(u+4*ib-RADIUS+0), (float)(v+blockDim.y-RADIUS));
                LocalBlock[SharedIdx+4*ib+1] = tex2D(tex,
                                                     (float)(u+4*ib-RADIUS+1), (float)(v+blockDim.y-RADIUS));
                LocalBlock[SharedIdx+4*ib+2] = tex2D(tex,
                                                     (float)(u+4*ib-RADIUS+2), (float)(v+blockDim.y-RADIUS));
                LocalBlock[SharedIdx+4*ib+3] = tex2D(tex,
                                                     (float)(u+4*ib-RADIUS+3), (float)(v+blockDim.y-RADIUS));
            }
        }

        __syncthreads();

        u >>= 2;    // index as uchar4 from here
        uchar4 *pSobel = (uchar4 *)(((char *) pSobelOriginal)+v*SobelPitch);
        SharedIdx = threadIdx.y * SharedPitch;

        for (ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x)
        {

            unsigned char pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+0];
            unsigned char pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+1];
            unsigned char pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+2];
            unsigned char pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+0];
            unsigned char pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+1];
            unsigned char pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+2];
            unsigned char pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+0];
            unsigned char pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+1];
            unsigned char pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+2];

            uchar4 out;

            out.x = ComputeSobel(pix00, pix01, pix02,
                                 pix10, pix11, pix12,
                                 pix20, pix21, pix22, fScale);

            pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+3];
            pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+3];
            pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+3];
            out.y = ComputeSobel(pix01, pix02, pix00,
                                 pix11, pix12, pix10,
                                 pix21, pix22, pix20, fScale);

            pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+4];
            pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+4];
            pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+4];
            out.z = ComputeSobel(pix02, pix00, pix01,
                                 pix12, pix10, pix11,
                                 pix22, pix20, pix21, fScale);

            pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+5];
            pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+5];
            pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+5];
            out.w = ComputeSobel(pix00, pix01, pix02,
                                 pix10, pix11, pix12,
                                 pix20, pix21, pix22, fScale);

            if (u+ib < w/4 && v < h)
            {
                pSobel[u+ib] = out;
            }
        }

        __syncthreads();
    }

    // copy the image as it is to the result
    __global__ void
    SobelCopyImage(Pixel *pSobelOriginal, unsigned int Pitch,
                   int w, int h, float fscale)
    {
        unsigned char *pSobel =
            (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

        for (int i = threadIdx.x; i < w; i += blockDim.x)
        {
            pSobel[i] = min(max((tex2D(tex, (float) i, (float) blockIdx.x) * fscale), 0.f), 255.f);
        }
    }

    // sobel filter application using texture memory
    __global__ void
    SobelTex(Pixel *pSobelOriginal, unsigned int Pitch,
             int w, int h, float fScale)
    {
        unsigned char *pSobel =
            (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

        for (int i = threadIdx.x; i < w; i += blockDim.x)
        {
            unsigned char pix00 = tex2D(tex, (float) i-1, (float) blockIdx.x-1);
            unsigned char pix01 = tex2D(tex, (float) i+0, (float) blockIdx.x-1);
            unsigned char pix02 = tex2D(tex, (float) i+1, (float) blockIdx.x-1);
            unsigned char pix10 = tex2D(tex, (float) i-1, (float) blockIdx.x+0);
            unsigned char pix11 = tex2D(tex, (float) i+0, (float) blockIdx.x+0);
            unsigned char pix12 = tex2D(tex, (float) i+1, (float) blockIdx.x+0);
            unsigned char pix20 = tex2D(tex, (float) i-1, (float) blockIdx.x+1);
            unsigned char pix21 = tex2D(tex, (float) i+0, (float) blockIdx.x+1);
            unsigned char pix22 = tex2D(tex, (float) i+1, (float) blockIdx.x+1);
            pSobel[i] = ComputeSobel(pix00, pix01, pix02,
                                     pix10, pix11, pix12,
                                     pix20, pix21, pix22, fScale);
        }
    }

    SobelFilter::SobelFilter()
    {
    }

    SobelFilter::~SobelFilter()
    {
        deleteTexture();
    }

    // initialize the cuda array with input image data
    void SobelFilter::setupTexture(int iw, int ih, Pixel *data, int Bpp)
    {
        hipChannelFormatDesc desc;

        if (Bpp == 1)
        {
            desc = hipCreateChannelDesc<unsigned char>();
        }
        else
        {
            desc = hipCreateChannelDesc<uchar4>();
        }

        checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
        checkCudaErrors(hipMemcpyToArray(array, 0, 0, data, Bpp*sizeof(Pixel)*iw*ih, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void **)&d_result, iw*ih*sizeof(Pixel)));
    }

    // free the cuda array
    void SobelFilter::deleteTexture()
    {
        checkCudaErrors(hipFreeArray(array));
        checkCudaErrors(hipFree(d_result));
    }

    // Wrapper for the __global__ call that sets up the texture and threads
    void SobelFilter::sobelFilter(Pixel *odata, int iw, int ih, enum SobelDisplayMode mode, float fScale)
    {
        checkCudaErrors(hipBindTextureToArray(tex, array));

        switch (mode)
        {
            case SOBELDISPLAY_IMAGE:
                SobelCopyImage<<<ih, 384>>>(d_result, iw, iw, ih, fScale);
                break;

            case SOBELDISPLAY_SOBELTEX:
                SobelTex<<<ih, 384>>>(d_result, iw, iw, ih, fScale);
                break;

            case SOBELDISPLAY_SOBELSHARED:
                {
                    dim3 threads(16,4);
                    int BlockWidth = 80; // must be divisible by 16 for coalescing
                    dim3 blocks = dim3(iw/(4*BlockWidth)+(0!=iw%(4*BlockWidth)), ih/threads.y+(0!=ih%threads.y));
                    int SharedPitch = ~0x3f&(4*(BlockWidth+2*RADIUS)+0x3f);
                    int sharedMem = SharedPitch*(threads.y+2*RADIUS);

                    // for the shared kernel, width must be divisible by 4
                    iw &= ~3;

                    SobelShared<<<blocks, threads, sharedMem>>>((uchar4 *) d_result,
                                                                iw,
                                                                BlockWidth, SharedPitch,
                                                                iw, ih, fScale);
                }
                break;
        }

        checkCudaErrors(hipUnbindTexture(tex));
        // write back final result to host memory
        checkCudaErrors(hipMemcpy(odata, d_result, iw*ih*sizeof(Pixel), hipMemcpyDeviceToHost));
    }
}

