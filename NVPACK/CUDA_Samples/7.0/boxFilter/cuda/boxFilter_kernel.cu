#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

#ifndef _BOXFILTER_KERNEL_CU_
#define _BOXFILTER_KERNEL_CU_

#include "boxFilter_kernel.h"
#include "hip/hip_vector_types.h"
#include "helper_functions.h"

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// texture memory is used to store the image data
texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
hipArray *d_array;

namespace gpu_bf
{
    // reads from 32-bit unsigned int array holding 8-bit RGBA

    // convert floating point rgba color to 32-bit integer
    __device__ unsigned int rgbaFloatToInt(float4 rgba)
    {
        rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
        rgba.y = __saturatef(rgba.y);
        rgba.z = __saturatef(rgba.z);
        rgba.w = __saturatef(rgba.w);
        return ((unsigned int)(rgba.w * 255.0f) << 24) |
               ((unsigned int)(rgba.z * 255.0f) << 16) |
               ((unsigned int)(rgba.y * 255.0f) <<  8) |
               ((unsigned int)(rgba.x * 255.0f));
    }

    __device__ float4 rgbaIntToFloat(unsigned int c)
    {
        float4 rgba;
        rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
        rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
        rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
        rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
        return rgba;
    }

    // row pass using texture memory reads
    __global__ void d_boxfilter_rgba_x(unsigned int *od, int w, int h, int r)
    {
        float scale = 1.0f / (float)((r << 1) + 1);
        unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

        // as long as address is always less than height, we do work
        if (y < h)
        {
            float4 t = make_float4(0.0f);

            for (int x = -r; x <= r; x++)
            {
                t += tex2D(rgbaTex, x, y);
            }

            od[y * w] = rgbaFloatToInt(t * scale);

            for (int x = 1; x < w; x++)
            {
                t += tex2D(rgbaTex, x + r, y);
                t -= tex2D(rgbaTex, x - r - 1, y);
                od[y * w + x] = rgbaFloatToInt(t * scale);
            }
        }
    }

    // column pass using coalesced global memory reads
    __global__ void d_boxfilter_rgba_y(unsigned int *id, unsigned int *od, int w, int h, int r)
    {
        unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
        id = &id[x];
        od = &od[x];

        float scale = 1.0f / (float)((r << 1) + 1);

        float4 t;
        // do left edge
        t = rgbaIntToFloat(id[0]) * r;

        for (int y = 0; y < (r + 1); y++)
        {
            t += rgbaIntToFloat(id[y*w]);
        }

        od[0] = rgbaFloatToInt(t * scale);

        for (int y = 1; y < (r + 1); y++)
        {
            t += rgbaIntToFloat(id[(y + r) * w]);
            t -= rgbaIntToFloat(id[0]);
            od[y * w] = rgbaFloatToInt(t * scale);
        }

        // main loop
        for (int y = (r + 1); y < (h - r); y++)
        {
            t += rgbaIntToFloat(id[(y + r) * w]);
            t -= rgbaIntToFloat(id[((y - r) * w) - w]);
            od[y * w] = rgbaFloatToInt(t * scale);
        }

        // do right edge
        for (int y = h - r; y < h; y++)
        {
            t += rgbaIntToFloat(id[(h - 1) * w]);
            t -= rgbaIntToFloat(id[((y - r) * w) - w]);
            od[y * w] = rgbaFloatToInt(t * scale);
        }
    }

    BoxFilterDemo::BoxFilterDemo()
    {
    }

    // free the allocated memory
    BoxFilterDemo::~BoxFilterDemo()
    {
        checkCudaErrors(hipFree(d_temp));
        checkCudaErrors(hipFree(d_result));
    }

    // initialize the texture with the input image array
    void BoxFilterDemo::init(int width, int height,const void *pImage)
    {
        // allocate memory to the intermediate arrays
        int size = width * height * sizeof(unsigned int);
        checkCudaErrors(hipMalloc(&d_temp, size));
        checkCudaErrors(hipMalloc(&d_result, size));

        // copy image data to array
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
        checkCudaErrors(hipMallocArray(&d_array, &channelDesc, width, height));
        checkCudaErrors(hipMemcpyToArray(d_array, 0, 0, pImage, size, hipMemcpyHostToDevice));
    }

    // apply Box filter
    void BoxFilterDemo::boxFilterRGBA(unsigned int *h_dest, int width, int height, int radius, int iterations, int nthreads)
    {
        // bind texture to the array containing input image data
        checkCudaErrors(hipBindTextureToArray(rgbaTex, d_array));

        for (int i=0; i<iterations; i++)
        {
            // sync host and start kernel computation timer_kernel
            checkCudaErrors(hipDeviceSynchronize());

            // use texture for horizontal pass
            d_boxfilter_rgba_x<<< height / nthreads, nthreads, 0 >>>(d_temp, width, height, radius);
            // use array for vertical paas
            d_boxfilter_rgba_y<<< width / nthreads, nthreads, 0 >>>(d_temp, d_result, width, height, radius);

            // sync host and stop computation timer_kernel
            checkCudaErrors(hipDeviceSynchronize());

            // copy back to the host memory
            checkCudaErrors(hipMemcpy(h_dest, d_result,  width * height * sizeof(unsigned int), hipMemcpyDeviceToHost));
        }
    }
}
#endif // #ifndef _BOXFILTER_KERNEL_CU_
