#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Recursive Gaussian filter
    sgreen 8/1/08

    This code sample implements a Gaussian blur using Deriche's recursive method:
    http://citeseer.ist.psu.edu/deriche93recursively.html

    This is similar to the box filter sample in the SDK, but it uses the previous
    outputs of the filter as well as the previous inputs. This is also known as an
    IIR (infinite impulse response) filter, since its response to an input impulse
    can last forever.

    The main advantage of this method is that the execution time is independent of
    the filter width.

    The GPU processes columns of the image in parallel. To avoid uncoalesced reads
    for the row pass we transpose the image and then transpose it back again
    afterwards.

    The implementation is based on code from the CImg library:
    http://cimg.sourceforge.net/
    Thanks to David Tschumperl� and all the CImg contributors!
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "recursiveGaussian.h"
#include "hip/hip_vector_types.h"

#include <android/log.h>

#define APP_NAME "CUDA_CORE"

#define LOGD(...) ((void)__android_log_print(ANDROID_LOG_DEBUG,  \
                                             APP_NAME, \
                                             __VA_ARGS__))
#define USE_SIMPLE_FILTER 0
#define BLOCK_DIM 16
#define CLAMP_TO_EDGE 1

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        LOGD("CUDA Runtime API error: %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
    {
        LOGD("CUDA error %s:%s", errorMessage, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

namespace gpu_recursive_gaussian
{

    // Round a / b to nearest higher integer value
    int iDivUp(int a, int b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }


    // Transpose a 2D array
    __global__ void d_transpose(uint *odata, uint *idata, int width, int height)
    {
        __shared__ uint block[BLOCK_DIM][BLOCK_DIM+1];

        // read the matrix tile into shared memory
        unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
        unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

        if ((xIndex < width) && (yIndex < height))
        {
            unsigned int index_in = yIndex * width + xIndex;
            block[threadIdx.y][threadIdx.x] = idata[index_in];
        }

        __syncthreads();

        // write the transposed matrix tile to global memory
        xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
        yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

        if ((xIndex < height) && (yIndex < width))
        {
            unsigned int index_out = yIndex * height + xIndex;
            odata[index_out] = block[threadIdx.x][threadIdx.y];
        }
    }

    // convert floating point rgba color to 32-bit integer
    __device__ uint rgbaFloatToInt(float4 rgba)
    {
        rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
        rgba.y = __saturatef(rgba.y);
        rgba.z = __saturatef(rgba.z);
        rgba.w = __saturatef(rgba.w);
        return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
    }

    // convert from 32-bit int to float4
    __device__ float4 rgbaIntToFloat(uint c)
    {
        float4 rgba;
        rgba.x = (c & 0xff) / 255.0f;
        rgba.y = ((c>>8) & 0xff) / 255.0f;
        rgba.z = ((c>>16) & 0xff) / 255.0f;
        rgba.w = ((c>>24) & 0xff) / 255.0f;
        return rgba;
    }

    /*
        simple 1st order recursive filter
        - processes one image column per thread

        parameters:
        id - pointer to input data (RGBA image packed into 32-bit integers)
        od - pointer to output data
        w  - image width
        h  - image height
        a  - blur parameter
    */

    __global__ void
    d_simpleRecursive_rgba(uint *id, uint *od, int w, int h, float a)
    {
        unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

        if (x >= w) return;

        id += x;    // advance pointers to correct column
        od += x;

        // forward pass
        float4 yp = rgbaIntToFloat(*id);  // previous output

        for (int y = 0; y < h; y++)
        {
            float4 xc = rgbaIntToFloat(*id);
            float4 yc = xc + a*(yp - xc);   // simple lerp between current and previous value
            *od = rgbaFloatToInt(yc);
            id += w;
            od += w;    // move to next row
            yp = yc;
        }

        // reset pointers to point to last element in column
        id -= w;
        od -= w;

        // reverse pass
        // ensures response is symmetrical
        yp = rgbaIntToFloat(*id);

        for (int y = h-1; y >= 0; y--)
        {
            float4 xc = rgbaIntToFloat(*id);
            float4 yc = xc + a*(yp - xc);
            *od = rgbaFloatToInt((rgbaIntToFloat(*od) + yc)*0.5f);
            id -= w;
            od -= w;  // move to previous row
            yp = yc;
        }
    }

    /*
        recursive Gaussian filter

        parameters:
        id - pointer to input data (RGBA image packed into 32-bit integers)
        od - pointer to output data
        w  - image width
        h  - image height
        a0-a3, b1, b2, coefp, coefn - filter parameters
    */

    __global__ void d_recursiveGaussian_rgba(uint *id, uint *od, int w, int h,
                                             float a0, float a1, float a2, float a3,
                                             float b1, float b2, float coefp, float coefn)
    {
        unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

        if (x >= w) return;

        id += x;    // advance pointers to correct column
        od += x;

        // forward pass
        float4 xp = make_float4(0.0f);  // previous input
        float4 yp = make_float4(0.0f);  // previous output
        float4 yb = make_float4(0.0f);  // previous output by 2
#if CLAMP_TO_EDGE
        xp = rgbaIntToFloat(*id);
        yb = coefp*xp;
        yp = yb;
#endif

        for (int y = 0; y < h; y++)
        {
            float4 xc = rgbaIntToFloat(*id);
            float4 yc = a0*xc + a1*xp - b1*yp - b2*yb;
            *od = rgbaFloatToInt(yc);
            id += w;
            od += w;    // move to next row
            xp = xc;
            yb = yp;
            yp = yc;
        }

        // reset pointers to point to last element in column
        id -= w;
        od -= w;

        // reverse pass
        // ensures response is symmetrical
        float4 xn = make_float4(0.0f);
        float4 xa = make_float4(0.0f);
        float4 yn = make_float4(0.0f);
        float4 ya = make_float4(0.0f);
#if CLAMP_TO_EDGE
        xn = xa = rgbaIntToFloat(*id);
        yn = coefn*xn;
        ya = yn;
#endif

        for (int y = h-1; y >= 0; y--)
        {
            float4 xc = rgbaIntToFloat(*id);
            float4 yc = a2*xn + a3*xa - b1*yn - b2*ya;
            xa = xn;
            xn = xc;
            ya = yn;
            yn = yc;
            *od = rgbaFloatToInt(rgbaIntToFloat(*od) + yc);
            id -= w;
            od -= w;  // move to previous row
        }
    }

    RecursiveGaussian::RecursiveGaussian()
    {
    }

    // free global memory
    RecursiveGaussian::~RecursiveGaussian()
    {
        checkCudaErrors(hipFree(d_img));
        checkCudaErrors(hipFree(d_temp));
        checkCudaErrors(hipFree(d_result));
    }

    // initialize global memory with input image data
    void RecursiveGaussian::init(int width, int height,uint *h_img)
    {
        unsigned int size = width * height * sizeof(unsigned int);

        // allocate device memory
        checkCudaErrors(hipMalloc((void **) &d_img, size));
        checkCudaErrors(hipMalloc((void **) &d_temp, size));
        checkCudaErrors(hipMalloc((void **) &d_result, size));
        checkCudaErrors(hipMemcpy(d_img, h_img, size, hipMemcpyHostToDevice));
    }

    // launch transpose kernel
    void RecursiveGaussian::transpose(uint *d_src, uint *d_dest, uint width, int height)
    {
        dim3 grid(iDivUp(width, BLOCK_DIM), iDivUp(height, BLOCK_DIM), 1);
        dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
        d_transpose<<< grid, threads >>>(d_dest, d_src, width, height);
        getLastCudaError("Kernel execution failed");
    }

    /*
        Perform Gaussian filter on a 2D image using CUDA

        Parameters:
        h_dest   - pointer to destination image in host memory
        width    - image width
        height   - image height
        sigma    - sigma of Gaussian
        order    - filter order (0, 1 or 2)
        nthreads - number of threads
    */

    void RecursiveGaussian::gaussianFilterRGBA(uint *h_dest, int width, int height,
                                               float sigma, int order, int nthreads)
    {
        // compute filter coefficients
        const float
        nsigma = sigma < 0.1f ? 0.1f : sigma,
        alpha = 1.695f / nsigma,
        ema = (float)std::exp(-alpha),
        ema2 = (float)std::exp(-2*alpha),
        b1 = -2*ema,
        b2 = ema2;

        float a0 = 0, a1 = 0, a2 = 0, a3 = 0, coefp = 0, coefn = 0;

        switch (order)
        {
            case 0:
                {
                    const float k = (1-ema)*(1-ema)/(1+2*alpha*ema-ema2);
                    a0 = k;
                    a1 = k*(alpha-1)*ema;
                    a2 = k*(alpha+1)*ema;
                    a3 = -k*ema2;
                }
                break;

            case 1:
                {
                    const float k = (1-ema)*(1-ema)/ema;
                    a0 = k*ema;
                    a1 = a3 = 0;
                    a2 = -a0;
                }
                break;

            case 2:
                {
                    const float
                    ea = (float)std::exp(-alpha),
                    k = -(ema2-1)/(2*alpha*ema),
                    kn = (-2*(-1+3*ea-3*ea*ea+ea*ea*ea)/(3*ea+1+3*ea*ea+ea*ea*ea));
                    a0 = kn;
                    a1 = -kn*(1+k*alpha)*ema;
                    a2 = kn*(1-k*alpha)*ema;
                    a3 = -kn*ema2;
                }
                break;

            default:
                LOGD("gaussianFilter: invalid order parameter!");
                return;
        }

        coefp = (a0+a1)/(1+b1+b2);
        coefn = (a2+a3)/(1+b1+b2);

        // process columns
#if USE_SIMPLE_FILTER
        d_simpleRecursive_rgba<<< iDivUp(width, nthreads), nthreads >>>(d_img, d_temp, width, height, ema);
#else
        d_recursiveGaussian_rgba<<< iDivUp(width, nthreads), nthreads >>>(d_img, d_temp, width, height, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
        getLastCudaError("Kernel execution failed");

        transpose(d_temp, d_result, width, height);
        getLastCudaError("transpose: Kernel execution failed");

        // process rows
#if USE_SIMPLE_FILTER
        d_simpleRecursive_rgba<<< iDivUp(height, nthreads), nthreads >>>(d_result, d_temp, height, width, ema);
#else
        d_recursiveGaussian_rgba<<< iDivUp(height, nthreads), nthreads >>>(d_result, d_temp, height, width, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
        getLastCudaError("Kernel execution failed");

        transpose(d_temp, d_result, height, width);

        // write back final result to host memory
        unsigned int size = width * height * sizeof(unsigned int);
        checkCudaErrors(hipMemcpy(h_dest, d_result, size, hipMemcpyDeviceToHost));
    }
}
