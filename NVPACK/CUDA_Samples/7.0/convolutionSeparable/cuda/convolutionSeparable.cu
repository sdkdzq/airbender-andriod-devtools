#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <assert.h>
#include "convolutionSeparable.h"
#include <android/log.h>

#define APP_NAME "CUDA_CORE"

#define LOGD(...) ((void)__android_log_print(ANDROID_LOG_DEBUG,  \
                                             APP_NAME, \
                                             __VA_ARGS__))

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        LOGD("CUDA Runtime API error: %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[2 * MAX_KERNEL_RADIUS + 1];

namespace gpu_convolution_separable
{
    // convert floating point rgba color to 32-bit integer
    __device__ unsigned int rgbaFloatToInt(float4 rgba)
    {
        rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
        rgba.y = __saturatef(rgba.y);
        rgba.z = __saturatef(rgba.z);
        rgba.w = __saturatef(rgba.w);
        return ((unsigned int)(rgba.w * 255.0f) << 24) |
               ((unsigned int)(rgba.z * 255.0f) << 16) |
               ((unsigned int)(rgba.y * 255.0f) <<  8) |
               ((unsigned int)(rgba.x * 255.0f));
    }

    __device__ float4 rgbaIntToFloat(unsigned int c)
    {
        float4 rgba;
        rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
        rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
        rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
        rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
        return rgba;
    }

    ////////////////////////////////////////////////////////////////////////////////
    // Row convolution filter
    ////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1

    __global__ void convolutionRowsKernel(
        unsigned int *d_Dst,
        unsigned int *d_Src,
        int imageW,
        int imageH,
        int pitch,
        int kernel_radius
    )
    {
        __shared__ unsigned int s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

        //Offset to the left halo edge
        const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
        const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

        d_Src += baseY * pitch + baseX;
        d_Dst += baseY * pitch + baseX;

        //Load main data
#pragma unroll

        for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
        {
            s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
        }

        //Load left halo
#pragma unroll

        for (int i = 0; i < ROWS_HALO_STEPS; i++)
        {
            s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
        }

        //Load right halo
#pragma unroll

        for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
        {
            s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
        }

        //Compute and store results
        __syncthreads();

#pragma unroll

        for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
        {
            float4 sum = { 0.0f, 0.0f, 0.0f, 0.0f };

            //#pragma unroll
            for (int j = -kernel_radius; j <= kernel_radius; j++)
            {
                float4 temp = rgbaIntToFloat(s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j]);

                sum.x += c_Kernel[kernel_radius - j] * temp.x;
                sum.y += c_Kernel[kernel_radius - j] * temp.y;
                sum.z += c_Kernel[kernel_radius - j] * temp.z;
                sum.w += c_Kernel[kernel_radius - j] * temp.w;
            }

            d_Dst[i * ROWS_BLOCKDIM_X] = rgbaFloatToInt(sum);
        }
    }

    ////////////////////////////////////////////////////////////////////////////////
    // Column convolution filter
    ////////////////////////////////////////////////////////////////////////////////
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 1

    __global__ void convolutionColumnsKernel(
        unsigned int *d_Dst,
        unsigned int *d_Src,
        int imageW,
        int imageH,
        int pitch,
        int kernel_radius
    )
    {
        __shared__ unsigned int s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

        //Offset to the upper halo edge
        const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
        const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
        d_Src += baseY * pitch + baseX;
        d_Dst += baseY * pitch + baseX;

        //Main data
#pragma unroll

        for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
        {
            s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
        }

        //Upper halo
#pragma unroll

        for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
        {
            s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
        }

        //Lower halo
#pragma unroll

        for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
        {
            s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
        }

        //Compute and store results
        __syncthreads();

        //#pragma unroll
        for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
        {
            float4 sum = { 0.0f, 0.0f, 0.0f, 0.0f };

            //#pragma unroll
            for (int j = -kernel_radius; j <= kernel_radius; j++)
            {
                float4 temp = rgbaIntToFloat(s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j]);
                sum.x += c_Kernel[kernel_radius - j] * temp.x;
                sum.y += c_Kernel[kernel_radius - j] * temp.y;
                sum.z += c_Kernel[kernel_radius - j] * temp.z;
                sum.w += c_Kernel[kernel_radius - j] * temp.w;
            }

            d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = rgbaFloatToInt(sum);
        }
    }


    ////////////////////////////////////////////////////////////////////////////////
    // ConvolutionSeparable Class implementation
    ////////////////////////////////////////////////////////////////////////////////

    ConvolutionSeparable::ConvolutionSeparable()
    {
    }

    ConvolutionSeparable::~ConvolutionSeparable()
    {
        // free device memory
        checkCudaErrors(hipFree(d_Input));
        checkCudaErrors(hipFree(d_Buffer));
        checkCudaErrors(hipFree(d_Output));
    }

    // Allocating and initializing CUDA global memory with input image data
    void ConvolutionSeparable::init(int width, int height,  unsigned int *hImage)
    {
        checkCudaErrors(hipMalloc((void **)&d_Input,   width * height * sizeof(unsigned int)));
        checkCudaErrors(hipMalloc((void **)&d_Output,  width * height * sizeof(unsigned int)));
        checkCudaErrors(hipMalloc((void **)&d_Buffer,  width * height * sizeof(unsigned int)));
        checkCudaErrors(hipMemcpy(d_Input, hImage,    width * height * sizeof(unsigned int), hipMemcpyHostToDevice));
    }

    // set the convolution kernel in constant memory
    void ConvolutionSeparable::setConvolutionKernel(float *h_Kernel)
    {
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, (2*MAX_KERNEL_RADIUS+1) * sizeof(float)));
    }

    // copy the final results back to the host memory
    void ConvolutionSeparable::copyToHost(unsigned int *processedData, int width, int height)
    {
        checkCudaErrors(hipMemcpy(processedData, d_Output, width * height * sizeof(unsigned int), hipMemcpyDeviceToHost));
    }

    // convolve with a column vector
    void ConvolutionSeparable::convolutionColumnsGPU(
        int imageW,
        int imageH,
        int kernel_radius
    )
    {
        assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
        assert(imageW % COLUMNS_BLOCKDIM_X == 0);
        assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

        dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
        dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

        convolutionColumnsKernel<<<blocks, threads>>>(
            d_Output,
            d_Buffer,
            imageW,
            imageH,
            imageW,
            kernel_radius
        );
        //    getLastCudaError("convolutionColumnsKernel() execution failed\n");
    }

    // convolve with a row vector
    void ConvolutionSeparable::convolutionRowsGPU(
        int imageW,
        int imageH,
        int kernel_radius
    )
    {
        assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
        assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
        assert(imageH % ROWS_BLOCKDIM_Y == 0);

        dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
        dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

        convolutionRowsKernel<<<blocks, threads>>>(
            d_Buffer,
            d_Input,
            imageW,
            imageH,
            imageW,
            kernel_radius
        );
        //    getLastCudaError("convolutionRowsKernel() execution failed\n");
    }
}
