
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256

__global__ void vadd(float* a, float* b, float* c) {
	int idx = threadIdx.x;
	if (idx < N) {
		c[idx] = a[idx] + b[idx];
	}
	return;
}

int main(int argc, char** argv) {
	float *ha, *hb, *hc, *da, *db, *dc;
	hipHostAlloc((void **)&ha, N * sizeof(float), hipHostMallocMapped);
	hipHostAlloc((void **)&hb, N * sizeof(float), hipHostMallocMapped);
	hipHostAlloc((void **)&hc, N * sizeof(float), hipHostMallocMapped);
	for (int i = 0; i < N; i ++) {
		ha[i] = 1.0;
		hb[i] = 2.0;
		hc[i] = 0.0;
	}
	hipHostGetDevicePointer((void **)&da, (void *)ha, 0);
	hipHostGetDevicePointer((void **)&db, (void *)hb, 0);
	hipHostGetDevicePointer((void **)&dc, (void *)hc, 0);
	vadd<<<1, N>>>(da, db, dc);
	hipDeviceSynchronize();
	for (int i = 0; i < N; i ++) {
		printf("%lf ", hc[i]);
	}
	printf("\n");
	return 0;
}
